#include "hip/hip_runtime.h"
// Equihash CUDA solver
// Copyright (c) 2016 John Tromp

#include "equi.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "blake2b.cu"

typedef uint16_t u16;
typedef uint64_t u64;

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

#ifndef RESTBITS
#define RESTBITS	4
#endif

// 2_log of number of buckets
#define BUCKBITS (DIGITBITS-RESTBITS)

#ifndef SAVEMEM
#if RESTBITS == 4
// can't save memory in such small buckets
#define SAVEMEM 1
#elif RESTBITS >= 8
// take advantage of law of large numbers (sum of 2^8 random numbers)
// this reduces (200,9) memory to under 144MB, with negligible discarding
#define SAVEMEM 9/14
#endif
#endif

// number of buckets
static const u32 NBUCKETS = 1<<BUCKBITS;
// bucket mask
static const u32 BUCKMASK = NBUCKETS-1;
// 2_log of number of slots per bucket
static const u32 SLOTBITS = RESTBITS+1+1;
static const u32 SLOTRANGE = 1<<SLOTBITS;
// number of slots per bucket
static const u32 NSLOTS = SLOTRANGE * SAVEMEM;
// number of per-xhash slots
static const u32 XFULL = 16;
// SLOTBITS mask
static const u32 SLOTMASK = SLOTRANGE-1;
// number of possible values of xhash (rest of n) bits
static const u32 NRESTS = 1<<RESTBITS;
// RESTBITS mask
static const u32 RESTMASK = NRESTS-1;
// number of blocks of hashes extracted from single 512 bit blake2b output
static const u32 NBLOCKS = (NHASHES+HASHESPERBLAKE-1)/HASHESPERBLAKE;
// nothing larger found in 100000 runs
static const u32 MAXSOLS = 8;

// tree node identifying its children as two different slots in
// a bucket on previous layer with the same rest bits (x-tra hash)
struct tree {
  u32 bid_s0_s1_x; // manual bitfields

  __device__ tree(const u32 idx, const u32 xh) {
    bid_s0_s1_x = idx << RESTBITS | xh;
  }
  __device__ tree(const u32 idx) {
    bid_s0_s1_x = idx;
  }
  __device__ tree(const u32 bid, const u32 s0, const u32 s1, const u32 xh) {
#ifdef XINTREE
  bid_s0_s1_x = ((((bid << SLOTBITS) | s0) << SLOTBITS) | s1) << RESTBITS | xh;
#else
  bid_s0_s1_x = (((bid << SLOTBITS) | s0) << SLOTBITS) | s1;
#endif
  }
  __device__ u32 getindex() const {
#ifdef XINTREE
    return bid_s0_s1_x >> RESTBITS;
#else
    return bid_s0_s1_x;
#endif
  }
  __device__ u32 bucketid() const {
#ifdef XINTREE
    return bid_s0_s1_x >> (2 * SLOTBITS + RESTBITS);
#else
    return bid_s0_s1_x >> (2 * SLOTBITS);
#endif
  }
  __device__ u32 slotid0() const {
#ifdef XINTREE
    return (bid_s0_s1_x >> SLOTBITS+RESTBITS) & SLOTMASK;
#else
    return (bid_s0_s1_x >> SLOTBITS) & SLOTMASK;
#endif
  }
  __device__ u32 slotid1() const {
#ifdef XINTREE
    return (bid_s0_s1_x >> RESTBITS) & SLOTMASK;
#else
    return bid_s0_s1_x & SLOTMASK;
#endif
  }
  __device__ u32 xhash() const {
    return bid_s0_s1_x & RESTMASK;
  }
};

union hashunit {
  u32 word;
  uchar bytes[sizeof(u32)];
};

#define WORDS(bits)     ((bits + 31) / 32)
#define HASHWORDS0 WORDS(WN - DIGITBITS + RESTBITS)
#define HASHWORDS1 WORDS(WN - 2*DIGITBITS + RESTBITS)

struct slot0 {
  tree attr;
  hashunit hash[HASHWORDS0];
};

struct slot1 {
  tree attr;
  hashunit hash[HASHWORDS1];
};

// a bucket is NSLOTS treenodes
typedef slot0 bucket0[NSLOTS];
typedef slot1 bucket1[NSLOTS];
// the N-bit hash consists of K+1 n-bit "digits"
// each of which corresponds to a layer of NBUCKETS buckets
typedef bucket0 digit0[NBUCKETS];
typedef bucket1 digit1[NBUCKETS];

// size (in bytes) of hash in round 0 <= r < WK
u32 hhashsize(const u32 r) {
#ifdef XINTREE
  const u32 hashbits = WN - (r+1) * DIGITBITS;
#else
  const u32 hashbits = WN - (r+1) * DIGITBITS + RESTBITS;
#endif
  return (hashbits + 7) / 8;
}
// size (in bytes) of hash in round 0 <= r < WK
__device__ u32 hashsize(const u32 r) {
#ifdef XINTREE
  const u32 hashbits = WN - (r+1) * DIGITBITS;
#else
  const u32 hashbits = WN - (r+1) * DIGITBITS + RESTBITS;
#endif
  return (hashbits + 7) / 8;
}

u32 hhashwords(u32 bytes) {
  return (bytes + 3) / 4;
}

__device__ u32 hashwords(u32 bytes) {
  return (bytes + 3) / 4;
}

// manages hash and tree data
struct htalloc {
  bucket0 *trees0[(WK+1)/2];
  bucket1 *trees1[WK/2];
};

typedef u32 bsizes[NBUCKETS];

struct equi {
  blake2b_state blake_ctx;
  htalloc hta;
  bsizes *nslots;
  proof *sols;
  u32 nsols;
  u32 nthreads;
  equi(const u32 n_threads) {
    nthreads = n_threads;
  }
  void setheadernonce(const char *headernonce, const u32 len) {
    setheader(&blake_ctx, headernonce);
    checkCudaErrors(hipMemset(nslots, 0, NBUCKETS * sizeof(u32)));
    nsols = 0;
  }
  __device__ u32 getnslots0(const u32 bid) {
    u32 &nslot = nslots[0][bid];
    const u32 n = min(nslot, NSLOTS);
    nslot = 0;
    return n;
  }
  __device__ u32 getnslots1(const u32 bid) {
    u32 &nslot = nslots[1][bid];
    const u32 n = min(nslot, NSLOTS);
    nslot = 0;
    return n;
  }
  __device__ void orderindices(u32 *indices, u32 size) {
    if (indices[0] > indices[size]) {
      for (u32 i=0; i < size; i++) {
        const u32 tmp = indices[i];
        indices[i] = indices[size+i];
        indices[size+i] = tmp;
      }
    }
  }
  __device__ void listindices1(const tree t, u32 *indices) {
    const bucket0 &buck = hta.trees0[0][t.bucketid()];
    const u32 size = 1 << 0;
    indices[0]    = buck[t.slotid0()].attr.getindex();
    indices[size] = buck[t.slotid1()].attr.getindex();
    orderindices(indices, size);
  }
  __device__ void listindices2(const tree t, u32 *indices) {
    const bucket1 &buck = hta.trees1[0][t.bucketid()];
    const u32 size = 1 << 1;
    listindices1(buck[t.slotid0()].attr, indices);
    listindices1(buck[t.slotid1()].attr, indices+size);
    orderindices(indices, size);
  }
  __device__ void listindices3(const tree t, u32 *indices) {
    const bucket0 &buck = hta.trees0[1][t.bucketid()];
    const u32 size = 1 << 2;
    listindices2(buck[t.slotid0()].attr, indices);
    listindices2(buck[t.slotid1()].attr, indices+size);
    orderindices(indices, size);
  }
  __device__ void listindices4(const tree t, u32 *indices) {
    const bucket1 &buck = hta.trees1[1][t.bucketid()];
    const u32 size = 1 << 3;
    listindices3(buck[t.slotid0()].attr, indices);
    listindices3(buck[t.slotid1()].attr, indices+size);
    orderindices(indices, size);
  }
  __device__ void listindices5(const tree t, u32 *indices) {
    const bucket0 &buck = hta.trees0[2][t.bucketid()];
    const u32 size = 1 << 4;
    listindices4(buck[t.slotid0()].attr, indices);
    listindices4(buck[t.slotid1()].attr, indices+size);
    orderindices(indices, size);
  }
  __device__ void listindices6(const tree t, u32 *indices) {
    const bucket1 &buck = hta.trees1[2][t.bucketid()];
    const u32 size = 1 << 5;
    listindices5(buck[t.slotid0()].attr, indices);
    listindices5(buck[t.slotid1()].attr, indices+size);
    orderindices(indices, size);
  }
  __device__ void listindices7(const tree t, u32 *indices) {
    const bucket0 &buck = hta.trees0[3][t.bucketid()];
    const u32 size = 1 << 6;
    listindices6(buck[t.slotid0()].attr, indices);
    listindices6(buck[t.slotid1()].attr, indices+size);
    orderindices(indices, size);
  }
  __device__ void listindices8(const tree t, u32 *indices) {
    const bucket1 &buck = hta.trees1[3][t.bucketid()];
    const u32 size = 1 << 7;
    listindices7(buck[t.slotid0()].attr, indices);
    listindices7(buck[t.slotid1()].attr, indices+size);
    orderindices(indices, size);
  }
  __device__ void listindices9(const tree t, u32 *indices) {
    const bucket0 &buck = hta.trees0[4][t.bucketid()];
    const u32 size = 1 << 8;
    listindices8(buck[t.slotid0()].attr, indices);
    listindices8(buck[t.slotid1()].attr, indices+size);
    orderindices(indices, size);
  }
  __device__ void candidate(const tree t) {
    proof prf;
#if WK==9
    listindices9(t, prf);
#elif WK==5
    listindices5(t, prf);
#else
#error not implemented
#endif
    if (probdupe(prf))
      return;
    u32 soli = atomicAdd(&nsols, 1);
    if (soli < MAXSOLS)
#if WK==9
      listindices9(t, sols[soli]);
#elif WK==5
      listindices5(t, sols[soli]);
#else
#error not implemented
#endif
  }
  void showbsizes(u32 r) {
#if defined(HIST) || defined(SPARK) || defined(LOGSPARK)
    u32 ns[NBUCKETS];
    checkCudaErrors(hipMemcpy(ns, nslots[r&1], NBUCKETS * sizeof(u32), hipMemcpyDeviceToHost));
    u32 binsizes[65];
    memset(binsizes, 0, 65 * sizeof(u32));
    for (u32 bucketid = 0; bucketid < NBUCKETS; bucketid++) {
      u32 bsize = min(ns[bucketid], NSLOTS) >> (SLOTBITS-6);
      binsizes[bsize]++;
    }
    for (u32 i=0; i < 65; i++) {
#ifdef HIST
      printf(" %d:%d", i, binsizes[i]);
#else
#ifdef SPARK
      u32 sparks = binsizes[i] / SPARKSCALE;
#else
      u32 sparks = 0;
      for (u32 bs = binsizes[i]; bs; bs >>= 1) sparks++;
      sparks = sparks * 7 / SPARKSCALE;
#endif
      printf("\342\226%c", '\201' + sparks);
#endif
    }
    printf("\n");
#endif
  }
  // proper dupe test is a little costly on GPU, so allow false negatives
  __device__ bool probdupe(u32 *prf) {
    unsigned short susp[PROOFSIZE];
    memset(susp, 0xffff, PROOFSIZE * sizeof(unsigned short));
    for (u32 i=0; i<PROOFSIZE; i++) {
      u32 bin = prf[i] & (PROOFSIZE-1);
      unsigned short msb = prf[i]>>WK;
      if (msb == susp[bin])
        return true;
      susp[bin] = msb;
    }
    return false;
  }
  struct htlayout {
    htalloc hta;
    u32 prevhashunits;
    u32 nexthashunits;
    u32 dunits;
    u32 prevbo;
    u32 nextbo;

    __device__ htlayout(equi *eq, u32 r): hta(eq->hta), prevhashunits(0), dunits(0) {
      u32 nexthashbytes = hashsize(r);
      nexthashunits = hashwords(nexthashbytes);
      prevbo = 0;
      nextbo = nexthashunits * sizeof(hashunit) - nexthashbytes; // 0-3
      if (r) {
        u32 prevhashbytes = hashsize(r-1);
        prevhashunits = hashwords(prevhashbytes);
        prevbo = prevhashunits * sizeof(hashunit) - prevhashbytes; // 0-3
        dunits = prevhashunits - nexthashunits;
      }
    }
    __device__ u32 getxhash0(const slot0* pslot) const {
#ifdef XINTREE
      return pslot->attr.xhash();
#elif WN == 200 && RESTBITS == 4
      return pslot->hash->bytes[prevbo] >> 4;
#elif WN == 200 && RESTBITS == 8
      return (pslot->hash->bytes[prevbo] & 0xf) << 4 | pslot->hash->bytes[prevbo+1] >> 4;
#elif WN == 144 && RESTBITS == 4
      return pslot->hash->bytes[prevbo] & 0xf;
#elif WN == 200 && RESTBITS == 6
      return (pslot->hash->bytes[prevbo] & 0x3) << 4 | pslot->hash->bytes[prevbo+1] >> 4;
#else
#error non implemented
#endif
    }
    __device__ u32 getxhash1(const slot1* pslot) const {
#ifdef XINTREE
      return pslot->attr.xhash();
#elif WN == 200 && RESTBITS == 4
      return pslot->hash->bytes[prevbo] & 0xf;
#elif WN == 200 && RESTBITS == 8
      return pslot->hash->bytes[prevbo];
#elif WN == 144 && RESTBITS == 4
      return pslot->hash->bytes[prevbo] & 0xf;
#elif WN == 200 && RESTBITS == 6
      return pslot->hash->bytes[prevbo] &0x3f;
#else
#error non implemented
#endif
    }
    __device__ bool equal(const hashunit *hash0, const hashunit *hash1) const {
      return hash0[prevhashunits-1].word == hash1[prevhashunits-1].word;
    }
  };

  struct collisiondata {
#ifdef XBITMAP
#if NSLOTS > 64
#error cant use XBITMAP with more than 64 slots
#endif
    u64 xhashmap[NRESTS];
    u64 xmap;
#else
#if RESTBITS <= 6
    typedef uchar xslot;
#else
    typedef u16 xslot;
#endif
    xslot nxhashslots[NRESTS];
    xslot xhashslots[NRESTS][XFULL];
    xslot *xx;
    u32 n0;
    u32 n1;
#endif
    u32 s0;

    __device__ void clear() {
#ifdef XBITMAP
      memset(xhashmap, 0, NRESTS * sizeof(u64));
#else
      memset(nxhashslots, 0, NRESTS * sizeof(xslot));
#endif
    }
    __device__ bool addslot(u32 s1, u32 xh) {
#ifdef XBITMAP
      xmap = xhashmap[xh];
      xhashmap[xh] |= (u64)1 << s1;
      s0 = ~0;
      return true;
#else
      n1 = (u32)nxhashslots[xh]++;
      if (n1 >= XFULL)
        return false;
      xx = xhashslots[xh];
      xx[n1] = s1;
      n0 = 0;
      return true;
#endif
    }
    __device__ bool nextcollision() const {
#ifdef XBITMAP
      return xmap != 0;
#else
      return n0 < n1;
#endif
    }
    __device__ u32 slot() {
#ifdef XBITMAP
      const u32 ffs = __ffsll(xmap);
      s0 += ffs; xmap >>= ffs;
      return s0;
#else
      return (u32)xx[n0++];
#endif
    }
  };
};

__global__ void digitH(equi *eq) {
  uchar hash[HASHOUT];
  blake2b_state state;
  equi::htlayout htl(eq, 0);
  const u32 hashbytes = hashsize(0);
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 block = id; block < NBLOCKS; block += eq->nthreads) {
    state = eq->blake_ctx;
    blake2b_gpu_hash(&state, block, hash, HASHOUT);
    for (u32 i = 0; i<HASHESPERBLAKE; i++) {
      const uchar *ph = hash + i * WN/8;
#if BUCKBITS == 16 && RESTBITS == 4
      const u32 bucketid = ((u32)ph[0] << 8) | ph[1];
#ifdef XINTREE
      const u32 xhash = ph[2] >> 4;
#endif
#elif BUCKBITS == 14 && RESTBITS == 6
      const u32 bucketid = ((u32)ph[0] << 6) | ph[1] >> 2;
#elif BUCKBITS == 12 && RESTBITS == 8
      const u32 bucketid = ((u32)ph[0] << 4) | ph[1] >> 4;
#elif BUCKBITS == 20 && RESTBITS == 4
      const u32 bucketid = ((((u32)ph[0] << 8) | ph[1]) << 4) | ph[2] >> 4;
#ifdef XINTREE
      const u32 xhash = ph[2] & 0xf;
#endif
#elif BUCKBITS == 12 && RESTBITS == 4
      const u32 bucketid = ((u32)ph[0] << 4) | ph[1] >> 4;
      const u32 xhash = ph[1] & 0xf;
#else
#error not implemented
#endif
      const u32 slot = atomicAdd(&eq->nslots[0][bucketid], 1);
      if (slot >= NSLOTS)
        continue;
      slot0 &s = eq->hta.trees0[0][bucketid][slot];
#ifdef XINTREE
      s.attr = tree(block*HASHESPERBLAKE+i, xhash);
#else
      s.attr = tree(block*HASHESPERBLAKE+i);
#endif
      memcpy(s.hash->bytes+htl.nextbo, ph+WN/8-hashbytes, hashbytes);
    }
  }
}

__global__ void digitO(equi *eq, const u32 r) {
  equi::htlayout htl(eq, r);
  equi::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot0 *buck = htl.hta.trees0[(r-1)/2][bucketid];
    u32 bsize = eq->getnslots0(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot0 *pslot1 = buck + s1;
      if (!cd.addslot(s1, htl.getxhash0(pslot1)))
        continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot0 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
          continue;
        u32 xorbucketid;
        u32 xhash;
        const uchar *bytes0 = pslot0->hash->bytes, *bytes1 = pslot1->hash->bytes;
#if WN == 200 && BUCKBITS == 16 && RESTBITS == 4 && defined(XINTREE)
        xorbucketid = ((((u32)(bytes0[htl.prevbo] ^ bytes1[htl.prevbo]) & 0xf) << 8)
                          | (bytes0[htl.prevbo+1] ^ bytes1[htl.prevbo+1])) << 4
                  | (xhash = bytes0[htl.prevbo+2] ^ bytes1[htl.prevbo+2]) >> 4;
        xhash &= 0xf;
#elif WN == 144 && BUCKBITS == 20 && RESTBITS == 4
        xorbucketid = ((((u32)(bytes0[htl.prevbo+1] ^ bytes1[htl.prevbo+1]) << 8)
                            | (bytes0[htl.prevbo+2] ^ bytes1[htl.prevbo+2])) << 4)
                    | (xhash = bytes0[htl.prevbo+3] ^ bytes1[htl.prevbo+3]) >> 4;
        xhash &= 0xf;
#elif WN == 96 && BUCKBITS == 12 && RESTBITS == 4
        xorbucketid = ((u32)(bytes0[htl.prevbo+1] ^ bytes1[htl.prevbo+1]) << 4)
                  | (xhash = bytes0[htl.prevbo+2] ^ bytes1[htl.prevbo+2]) >> 4;
        xhash &= 0xf;
#elif WN == 200 && BUCKBITS == 14 && RESTBITS == 6
        xorbucketid = ((((u32)(bytes0[htl.prevbo+1] ^ bytes1[htl.prevbo+1]) & 0xf) << 8)
                           | (bytes0[htl.prevbo+2] ^ bytes1[htl.prevbo+2])) << 2
                           | (bytes0[htl.prevbo+3] ^ bytes1[htl.prevbo+3]) >> 6;
#else
#error not implemented
#endif
        const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot1 &xs = htl.hta.trees1[r/2][xorbucketid][xorslot];
#ifdef XINTREE
        xs.attr = tree(bucketid, s0, s1, xhash);
#else
        xs.attr = tree(bucketid, s0, s1);
#endif
        for (u32 i=htl.dunits; i < htl.prevhashunits; i++)
          xs.hash[i-htl.dunits].word = pslot0->hash[i].word ^ pslot1->hash[i].word;
      }
    }
  }
}

__global__ void digitE(equi *eq, const u32 r) {
  equi::htlayout htl(eq, r);
  equi::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot1 *buck = htl.hta.trees1[(r-1)/2][bucketid];
    u32 bsize = eq->getnslots1(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot1 *pslot1 = buck + s1;
      if (!cd.addslot(s1, htl.getxhash1(pslot1)))
        continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot1 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
          continue;
        u32 xorbucketid;
        u32 xhash;
        const uchar *bytes0 = pslot0->hash->bytes, *bytes1 = pslot1->hash->bytes;
#if WN == 200 && BUCKBITS == 16 && RESTBITS == 4 && defined(XINTREE)
        xorbucketid = ((u32)(bytes0[htl.prevbo] ^ bytes1[htl.prevbo]) << 8)
                        | (bytes0[htl.prevbo+1] ^ bytes1[htl.prevbo+1]);
                  xhash = (bytes0[htl.prevbo+2] ^ bytes1[htl.prevbo+2]) >> 4;
#elif WN == 144 && BUCKBITS == 20 && RESTBITS == 4
        xorbucketid = ((((u32)(bytes0[htl.prevbo+1] ^ bytes1[htl.prevbo+1]) << 8)
                            | (bytes0[htl.prevbo+2] ^ bytes1[htl.prevbo+2])) << 4)
                            | (bytes0[htl.prevbo+3] ^ bytes1[htl.prevbo+3]) >> 4;
#elif WN == 96 && BUCKBITS == 12 && RESTBITS == 4
        xorbucketid = ((u32)(bytes0[htl.prevbo+1] ^ bytes1[htl.prevbo+1]) << 4)
                          | (bytes0[htl.prevbo+2] ^ bytes1[htl.prevbo+2]) >> 4;
#elif WN == 200 && BUCKBITS == 14 && RESTBITS == 6
        xorbucketid = ((u32)(bytes0[htl.prevbo+1] ^ bytes1[htl.prevbo+1]) << 6)
                          | (bytes0[htl.prevbo+2] ^ bytes1[htl.prevbo+2]) >> 2;
#else
#error not implemented
#endif
        const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot0 &xs = htl.hta.trees0[r/2][xorbucketid][xorslot];
#ifdef XINTREE
        xs.attr = tree(bucketid, s0, s1, xhash);
#else
        xs.attr = tree(bucketid, s0, s1);
#endif
        for (u32 i=htl.dunits; i < htl.prevhashunits; i++)
          xs.hash[i-htl.dunits].word = pslot0->hash[i].word ^ pslot1->hash[i].word;
      }
    }
  }
}

#ifdef UNROLL
__global__ void digit_1(equi *eq) {
  equi::htlayout htl(eq, 1);
  equi::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot0 *buck = htl.hta.trees0[0][bucketid];
    u32 bsize = eq->getnslots0(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot0 *pslot1 = buck + s1;
      if (!cd.addslot(s1, htl.getxhash0(pslot1)))
        continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot0 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
          continue;
        const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
        const u32 bexor = __byte_perm(xor0, 0, 0x0123);
        const u32 xorbucketid = bexor >> 4 & BUCKMASK;
        const u32 xhash = bexor & 0xf;
        const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot1 &xs = htl.hta.trees1[0][xorbucketid][xorslot];
        xs.attr = tree(bucketid, s0, s1, xhash);
        xs.hash[0].word = pslot0->hash[1].word ^ pslot1->hash[1].word;
        xs.hash[1].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
        xs.hash[2].word = pslot0->hash[3].word ^ pslot1->hash[3].word;
        xs.hash[3].word = pslot0->hash[4].word ^ pslot1->hash[4].word;
        xs.hash[4].word = pslot0->hash[5].word ^ pslot1->hash[5].word;
      }
    }
  }
}
__global__ void digit2(equi *eq) {
  equi::htlayout htl(eq, 2);
  equi::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot1 *buck = htl.hta.trees1[0][bucketid];
    u32 bsize = eq->getnslots1(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot1 *pslot1 = buck + s1;
      if (!cd.addslot(s1, htl.getxhash1(pslot1)))
        continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot1 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
          continue;
        const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
        const u32 bexor = __byte_perm(xor0, 0, 0x0123);
        const u32 xorbucketid = bexor >> 16;
        const u32 xhash = bexor >> 12 & 0xf;
        const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot0 &xs = htl.hta.trees0[1][xorbucketid][xorslot];
        xs.attr = tree(bucketid, s0, s1, xhash);
        xs.hash[0].word = xor0;
        xs.hash[1].word = pslot0->hash[1].word ^ pslot1->hash[1].word;
        xs.hash[2].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
        xs.hash[3].word = pslot0->hash[3].word ^ pslot1->hash[3].word;
        xs.hash[4].word = pslot0->hash[4].word ^ pslot1->hash[4].word;
      }
    }
  }
}
__global__ void digit3(equi *eq) {
  equi::htlayout htl(eq, 3);
  equi::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot0 *buck = htl.hta.trees0[1][bucketid];
    u32 bsize = eq->getnslots0(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot0 *pslot1 = buck + s1;
      if (!cd.addslot(s1, htl.getxhash0(pslot1)))
        continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot0 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
          continue;
        const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
        const u32 xor1 = pslot0->hash[1].word ^ pslot1->hash[1].word;
        const u32 bexor = __byte_perm(xor0, xor1, 0x1234);
        const u32 xorbucketid = bexor >> 4 & BUCKMASK;
        const u32 xhash = bexor & 0xf;
        const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot1 &xs = htl.hta.trees1[1][xorbucketid][xorslot];
        xs.attr = tree(bucketid, s0, s1, xhash);
        xs.hash[0].word = xor1;
        xs.hash[1].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
        xs.hash[2].word = pslot0->hash[3].word ^ pslot1->hash[3].word;
        xs.hash[3].word = pslot0->hash[4].word ^ pslot1->hash[4].word;
      }
    }
  }
}
__global__ void digit4(equi *eq) {
  equi::htlayout htl(eq, 4);
  equi::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot1 *buck = htl.hta.trees1[1][bucketid];
    u32 bsize = eq->getnslots1(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot1 *pslot1 = buck + s1;
      if (!cd.addslot(s1, htl.getxhash1(pslot1)))
        continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot1 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
          continue;
        const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
        const u32 bexor = __byte_perm(xor0, 0, 0x4123);
        const u32 xorbucketid = bexor >> 8;
        const u32 xhash = bexor >> 4 & 0xf;
        const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot0 &xs = htl.hta.trees0[2][xorbucketid][xorslot];
        xs.attr = tree(bucketid, s0, s1, xhash);
        xs.hash[0].word = xor0;
        xs.hash[1].word = pslot0->hash[1].word ^ pslot1->hash[1].word;
        xs.hash[2].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
        xs.hash[3].word = pslot0->hash[3].word ^ pslot1->hash[3].word;
      }
    }
  }
}
__global__ void digit5(equi *eq) {
  equi::htlayout htl(eq, 5);
  equi::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot0 *buck = htl.hta.trees0[2][bucketid];
    u32 bsize = eq->getnslots0(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot0 *pslot1 = buck + s1;
      if (!cd.addslot(s1, htl.getxhash0(pslot1)))
        continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot0 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
          continue;
        const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
        const u32 xor1 = pslot0->hash[1].word ^ pslot1->hash[1].word;
        const u32 bexor = __byte_perm(xor0, xor1, 0x2345);
        const u32 xorbucketid = bexor >> 4 & BUCKMASK;
        const u32 xhash = bexor & 0xf;
        const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot1 &xs = htl.hta.trees1[2][xorbucketid][xorslot];
        xs.attr = tree(bucketid, s0, s1, xhash);
        xs.hash[0].word = xor1;
        xs.hash[1].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
        xs.hash[2].word = pslot0->hash[3].word ^ pslot1->hash[3].word;
      }
    }
  }
}
__global__ void digit6(equi *eq) {
  equi::htlayout htl(eq, 6);
  equi::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot1 *buck = htl.hta.trees1[2][bucketid];
    u32 bsize = eq->getnslots1(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot1 *pslot1 = buck + s1;
      if (!cd.addslot(s1, htl.getxhash1(pslot1)))
        continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot1 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
          continue;
        const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
        const u32 xor1 = pslot0->hash[1].word ^ pslot1->hash[1].word;
        const u32 bexor = __byte_perm(xor0, xor1, 0x2345);
        const u32 xorbucketid = bexor >> 16;
        const u32 xhash = bexor >> 12 & 0xf;
        const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot0 &xs = htl.hta.trees0[3][xorbucketid][xorslot];
        xs.attr = tree(bucketid, s0, s1, xhash);
        xs.hash[0].word = xor1;
        xs.hash[1].word = pslot0->hash[2].word ^ pslot1->hash[2].word;
      }
    }
  }
}
__global__ void digit7(equi *eq) {
  equi::htlayout htl(eq, 7);
  equi::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot0 *buck = htl.hta.trees0[3][bucketid];
    u32 bsize = eq->getnslots0(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot0 *pslot1 = buck + s1;
      if (!cd.addslot(s1, htl.getxhash0(pslot1)))
        continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot0 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
          continue;
        const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
        const u32 bexor = __byte_perm(xor0, 0, 0x4012);
        const u32 xorbucketid = bexor >> 4 & BUCKMASK;
        const u32 xhash = bexor & 0xf;
        const u32 xorslot = atomicAdd(&eq->nslots[1][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot1 &xs = htl.hta.trees1[3][xorbucketid][xorslot];
        xs.attr = tree(bucketid, s0, s1, xhash);
        xs.hash[0].word = xor0;
        xs.hash[1].word = pslot0->hash[1].word ^ pslot1->hash[1].word;
      }
    }
  }
}
__global__ void digit8(equi *eq) {
  equi::htlayout htl(eq, 8);
  equi::collisiondata cd;
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid=id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot1 *buck = htl.hta.trees1[3][bucketid];
    u32 bsize = eq->getnslots1(bucketid);
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot1 *pslot1 = buck + s1;
      if (!cd.addslot(s1, htl.getxhash1(pslot1)))
        continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot1 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash))
          continue;
        const u32 xor0 = pslot0->hash->word ^ pslot1->hash->word;
        const u32 xor1 = pslot0->hash[1].word ^ pslot1->hash[1].word;
        const u32 bexor = __byte_perm(xor0, xor1, 0x3456);
        const u32 xorbucketid = bexor >> 16;
        const u32 xhash = bexor >> 12 & 0xf;
        const u32 xorslot = atomicAdd(&eq->nslots[0][xorbucketid], 1);
        if (xorslot >= NSLOTS)
          continue;
        slot0 &xs = htl.hta.trees0[4][xorbucketid][xorslot];
        xs.attr = tree(bucketid, s0, s1, xhash);
        xs.hash[0].word = xor1;
      }
    }
  }
}
#endif

__global__ void digitK(equi *eq) {
  equi::collisiondata cd;
  equi::htlayout htl(eq, WK);
  const u32 id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 bucketid = id; bucketid < NBUCKETS; bucketid += eq->nthreads) {
    cd.clear();
    slot0 *buck = htl.hta.trees0[(WK-1)/2][bucketid];
    u32 bsize = eq->getnslots0(bucketid); // assume WK odd
    for (u32 s1 = 0; s1 < bsize; s1++) {
      const slot0 *pslot1 = buck + s1;
      if (!cd.addslot(s1, htl.getxhash0(pslot1))) // assume WK odd
        continue;
      for (; cd.nextcollision(); ) {
        const u32 s0 = cd.slot();
        const slot0 *pslot0 = buck + s0;
        if (htl.equal(pslot0->hash, pslot1->hash)) {
#ifdef XINTREE
          eq->candidate(tree(bucketid, s0, s1, 0));
#else
          eq->candidate(tree(bucketid, s0, s1));
#endif
        }
      }
    }
  }
}

#include <unistd.h>

int main(int argc, char **argv) {
  int nthreads = 8192;
  int nonce = 0;
  int tpb = 0;
  int range = 1;
  bool showsol = false;
  const char *header = "";
  int c;
  while ((c = getopt (argc, argv, "h:n:r:t:p:s")) != -1) {
    switch (c) {
      case 'h':
        header = optarg;
        break;
      case 'n':
        nonce = atoi(optarg);
        break;
      case 't':
        nthreads = atoi(optarg);
        break;
      case 'p':
        tpb = atoi(optarg);
        break;
      case 'r':
        range = atoi(optarg);
        break;
      case 's':
        showsol = true;
        break;
    }
  }
  if (!tpb) // if not set, then default threads per block to roughly square root of threads
    for (tpb = 1; tpb*tpb < nthreads; tpb *= 2) ;

  printf("Looking for wagner-tree on (\"%s\",%d", header, nonce);
  if (range > 1)
    printf("-%d", nonce+range-1);
  printf(") with %d %d-bits digits and %d threads (%d per block)\n", NDIGITS, DIGITBITS, nthreads, tpb);
  equi eq(nthreads);

  char headernonce[HEADERNONCELEN];
  u32 hdrlen = strlen(header);
  memcpy(headernonce, header, hdrlen);
  memset(headernonce+hdrlen, 0, sizeof(headernonce)-hdrlen);

  u32 *heap0, *heap1;
  checkCudaErrors(hipMalloc((void**)&heap0, sizeof(digit0)));
  checkCudaErrors(hipMalloc((void**)&heap1, sizeof(digit1)));
  for (u32 r=0; r < WK; r++)
    if ((r&1) == 0)
      eq.hta.trees0[r/2]  = (bucket0 *)(heap0 + r/2);
    else
      eq.hta.trees1[r/2]  = (bucket1 *)(heap1 + r/2);

  checkCudaErrors(hipMalloc((void**)&eq.nslots, 2 * NBUCKETS * sizeof(u32)));
  checkCudaErrors(hipMalloc((void**)&eq.sols, MAXSOLS * sizeof(proof)));

  equi *device_eq;
  checkCudaErrors(hipMalloc((void**)&device_eq, sizeof(equi)));

  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  proof sols[MAXSOLS];
  u32 sumnsols = 0;
  for (int r = 0; r < range; r++) {
    hipEventRecord(start, NULL);
    ((u32 *)headernonce)[32] = htole32(nonce+r);
    eq.setheadernonce(headernonce, sizeof(headernonce));
    checkCudaErrors(hipMemcpy(device_eq, &eq, sizeof(equi), hipMemcpyHostToDevice));
    printf("Digit 0\n");
    digitH<<<nthreads/tpb,tpb >>>(device_eq);
    eq.showbsizes(0);
#if BUCKBITS == 16 && RESTBITS == 4 && defined XINTREE && defined(UNROLL)
    printf("Digit %d\n", 1);
    digit_1<<<nthreads/tpb,tpb >>>(device_eq);
    eq.showbsizes(1);
    printf("Digit %d\n", 2);
    digit2<<<nthreads/tpb,tpb >>>(device_eq);
    eq.showbsizes(2);
    printf("Digit %d\n", 3);
    digit3<<<nthreads/tpb,tpb >>>(device_eq);
    eq.showbsizes(3);
    printf("Digit %d\n", 4);
    digit4<<<nthreads/tpb,tpb >>>(device_eq);
    eq.showbsizes(4);
    printf("Digit %d\n", 5);
    digit5<<<nthreads/tpb,tpb >>>(device_eq);
    eq.showbsizes(5);
    printf("Digit %d\n", 6);
    digit6<<<nthreads/tpb,tpb >>>(device_eq);
    eq.showbsizes(6);
    printf("Digit %d\n", 7);
    digit7<<<nthreads/tpb,tpb >>>(device_eq);
    eq.showbsizes(7);
    printf("Digit %d\n", 8);
    digit8<<<nthreads/tpb,tpb >>>(device_eq);
    eq.showbsizes(8);
#else
    for (u32 r=1; r < WK; r++) {
      printf("Digit %d\n", r);
      r&1 ?  digitO<<<nthreads/tpb,tpb >>>(device_eq, r)
          :  digitE<<<nthreads/tpb,tpb >>>(device_eq, r);
      eq.showbsizes(r);
    }
#endif
    printf("Digit %d\n", WK);
    digitK<<<nthreads/tpb,tpb >>>(device_eq);

    checkCudaErrors(hipMemcpy(&eq, device_eq, sizeof(equi), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(sols, eq.sols, MAXSOLS * sizeof(proof), hipMemcpyDeviceToHost));
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    float duration;
    hipEventElapsedTime(&duration, start, stop);
      printf("%d rounds completed in %.3f seconds.\n", WK, duration / 1000.0f);

    u32 nsols = 0;
    for (unsigned s = 0; s < eq.nsols; s++) {
      if (duped(sols[s])) {
        printf("Duped!\n");
        continue;
      }
      nsols++;
      if (showsol) {
        printf("Solution");
        for (int i = 0; i < PROOFSIZE; i++)
          printf(" %jx", (uintmax_t)sols[s][i]);
        printf("\n");
      }
    }
    printf("%d solutions\n", nsols);
    sumnsols += nsols;
  }
  checkCudaErrors(hipFree(eq.nslots));
  checkCudaErrors(hipFree(eq.sols));
  checkCudaErrors(hipFree(eq.hta.trees0[0]));
  checkCudaErrors(hipFree(eq.hta.trees1[0]));

  printf("%d total solutions\n", sumnsols);
  return 0;
}
