#include "hip/hip_runtime.h"
extern "C"
{
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
}

#include "miner.h"

#include "cuda_helper.h"
#include "cuda_quark.h"

#include <stdio.h>

static uint32_t *d_hash;

// Speicher zur Generierung der Noncevektoren für die bedingten Hashes
static uint32_t *d_branch1Nonces;
static uint32_t *d_branch2Nonces;
static uint32_t *d_branch3Nonces;

// Original Quarkhash Funktion aus einem miner Quelltext
extern "C" void quarkhash(void *state, const void *input)
{
	unsigned char hash[64];

	sph_blake512_context ctx_blake;
	sph_bmw512_context ctx_bmw;
	sph_groestl512_context ctx_groestl;
	sph_jh512_context ctx_jh;
	sph_keccak512_context ctx_keccak;
	sph_skein512_context ctx_skein;

	sph_blake512_init(&ctx_blake);
	sph_blake512 (&ctx_blake, input, 80);
	sph_blake512_close(&ctx_blake, (void*) hash);

	sph_bmw512_init(&ctx_bmw);
	sph_bmw512 (&ctx_bmw, (const void*) hash, 64);
	sph_bmw512_close(&ctx_bmw, (void*) hash);

	if (hash[0] & 0x8)
	{
		sph_groestl512_init(&ctx_groestl);
		sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
		sph_groestl512_close(&ctx_groestl, (void*) hash);
	}
	else
	{
		sph_skein512_init(&ctx_skein);
		sph_skein512 (&ctx_skein, (const void*) hash, 64);
		sph_skein512_close(&ctx_skein, (void*) hash);
	}

	sph_groestl512_init(&ctx_groestl);
	sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
	sph_groestl512_close(&ctx_groestl, (void*) hash);

	sph_jh512_init(&ctx_jh);
	sph_jh512 (&ctx_jh, (const void*) hash, 64);
	sph_jh512_close(&ctx_jh, (void*) hash);

	if (hash[0] & 0x8)
	{
		sph_blake512_init(&ctx_blake);
		sph_blake512 (&ctx_blake, (const void*) hash, 64);
		sph_blake512_close(&ctx_blake, (void*) hash);
	}
	else
	{
		sph_bmw512_init(&ctx_bmw);
		sph_bmw512 (&ctx_bmw, (const void*) hash, 64);
		sph_bmw512_close(&ctx_bmw, (void*) hash);
	}

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
	sph_keccak512_close(&ctx_keccak, (void*) hash);

	sph_skein512_init(&ctx_skein);
	sph_skein512 (&ctx_skein, (const void*) hash, 64);
	sph_skein512_close(&ctx_skein, (void*) hash);

	if (hash[0] & 0x8)
	{
		sph_keccak512_init(&ctx_keccak);
		sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
		sph_keccak512_close(&ctx_keccak, (void*) hash);
	}
	else
	{
		sph_jh512_init(&ctx_jh);
		sph_jh512 (&ctx_jh, (const void*) hash, 64);
		sph_jh512_close(&ctx_jh, (void*) hash);
	}

	memcpy(state, hash, 32);
}

#ifdef _DEBUG
#define TRACE(algo) { \
	if (max_nonce == 1 && pdata[19] <= 1) { \
		uint32_t* debugbuf = NULL; \
		hipHostMalloc(&debugbuf, 32); \
		hipMemcpy(debugbuf, d_hash[thr_id], 32, hipMemcpyDeviceToHost); \
		printf("quark %s %08x %08x %08x %08x...%08x... \n", algo, swab32(debugbuf[0]), swab32(debugbuf[1]), \
			swab32(debugbuf[2]), swab32(debugbuf[3]), swab32(debugbuf[7])); \
		hipHostFree(debugbuf); \
	} \
}
#else
#define TRACE(algo) {}
#endif

static bool init_flag = false;

extern "C" int scanhash_quark(struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t endiandata[20];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];

	uint32_t def_thr = 1U << 10; // 256*4096
	uint32_t throughput = def_thr;
	if (init_flag) throughput = min(throughput, max_nonce - first_nonce);

	ptarget[7] = 0x00F;

	if (!init_flag)
	{
		// gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		hipGetLastError();
		CUDA_SAFE_CALL(hipMalloc(&d_hash, (size_t) 64 * throughput));

		quark_blake512_cpu_init(throughput);
		quark_groestl512_cpu_init(throughput);
		quark_skein512_cpu_init(throughput);
		quark_bmw512_cpu_init(throughput);
		quark_keccak512_cpu_init(throughput);
		quark_jh512_cpu_init(throughput);
		// quark_compactTest_cpu_init(throughput);

		// if (cuda_arch[dev_id] >= 300) {
			hipMalloc(&d_branch1Nonces, sizeof(uint32_t)*throughput);
			hipMalloc(&d_branch2Nonces, sizeof(uint32_t)*throughput);
			hipMalloc(&d_branch3Nonces, sizeof(uint32_t)*throughput);
		/*
		} else {
			hipMalloc(&d_hash_br2[thr_id], (size_t) 64 * throughput);
		}
		*/

		cuda_check_cpu_init(throughput);
		CUDA_SAFE_CALL(hipGetLastError());

		init_flag = true;
	}

	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	quark_blake512_cpu_setBlock_80(endiandata);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;
		uint32_t nrm1=0, nrm2=0, nrm3=0;

		quark_blake512_cpu_hash_80(throughput, pdata[19], d_hash); order++;
		TRACE("blake  :");
		quark_bmw512_cpu_hash_64(throughput, pdata[19], NULL, d_hash, order++);
		TRACE("bmw    :");

		// if (cuda_arch[dev_id] >= 300) {

			/*quark_compactTest_single_false_cpu_hash_64(throughput, pdata[19], d_hash, NULL,
				d_branch3Nonces, &nrm3, order++);*/

			// nur den Skein Branch weiterverfolgen
			quark_skein512_cpu_hash_64(nrm3, pdata[19], d_branch3Nonces, d_hash, order++);

			// das ist der unbedingte Branch für Groestl512
			quark_groestl512_cpu_hash_64(nrm3, pdata[19], d_branch3Nonces, d_hash, order++);

			// das ist der unbedingte Branch für JH512
			quark_jh512_cpu_hash_64(nrm3, pdata[19], d_branch3Nonces, d_hash, order++);

			// quarkNonces in branch1 und branch2 aufsplitten gemäss if (hash[0] & 0x8)
			/*quark_compactTest_cpu_hash_64(nrm3, pdata[19], d_hash, d_branch3Nonces,
				d_branch1Nonces, &nrm1,
				d_branch2Nonces, &nrm2,
				order++);*/

			// das ist der bedingte Branch für Blake512
			quark_blake512_cpu_hash_64(nrm1, pdata[19], d_branch1Nonces, d_hash, order++);

			// das ist der bedingte Branch für Bmw512
			quark_bmw512_cpu_hash_64(nrm2, pdata[19], d_branch2Nonces, d_hash, order++);

			// das ist der unbedingte Branch für Keccak512
			quark_keccak512_cpu_hash_64(nrm3, pdata[19], d_branch3Nonces, d_hash, order++);

			// das ist der unbedingte Branch für Skein512
			quark_skein512_cpu_hash_64(nrm3, pdata[19], d_branch3Nonces, d_hash, order++);

			// quarkNonces in branch1 und branch2 aufsplitten gemäss if (hash[0] & 0x8)
			/*quark_compactTest_cpu_hash_64(nrm3, pdata[19], d_hash, d_branch3Nonces,
				d_branch1Nonces, &nrm1,
				d_branch2Nonces, &nrm2,
				order++);*/

			quark_keccak512_cpu_hash_64(nrm1, pdata[19], d_branch1Nonces, d_hash, order++);
			quark_jh512_cpu_hash_64(nrm2, pdata[19], d_branch2Nonces, d_hash, order++);

			work->nonces[0] = cuda_check_hash_branch(nrm3, pdata[19], d_branch3Nonces, d_hash, order++);
			work->nonces[1] = 0;
		
		/* } else {
			// algo permutations are made with 2 different buffers 

			quark_filter_cpu_sm2(thr_id, throughput, d_hash[thr_id], d_hash_br2[thr_id]);
			quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash_br2[thr_id], order++);
			quark_merge_cpu_sm2(thr_id, throughput, d_hash[thr_id], d_hash_br2[thr_id]);
			TRACE("perm1  :");

			quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			TRACE("groestl:");
			quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			TRACE("jh512  :");

			quark_filter_cpu_sm2(thr_id, throughput, d_hash[thr_id], d_hash_br2[thr_id]);
			quark_blake512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash_br2[thr_id], order++);
			quark_merge_cpu_sm2(thr_id, throughput, d_hash[thr_id], d_hash_br2[thr_id]);
			TRACE("perm2  :");

			quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			TRACE("keccak :");
			quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			TRACE("skein  :");

			quark_filter_cpu_sm2(thr_id, throughput, d_hash[thr_id], d_hash_br2[thr_id]);
			quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
			quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash_br2[thr_id], order++);
			quark_merge_cpu_sm2(thr_id, throughput, d_hash[thr_id], d_hash_br2[thr_id]);
			TRACE("perm3  :");

			CUDA_LOG_ERROR();
			work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
			work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
		}
		*/

		*hashes_done = pdata[19] - first_nonce + throughput;

		if (work->nonces[0] != UINT32_MAX)
		{
			uint32_t vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);
			quarkhash(vhash, endiandata);

			if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				// work_set_target_ratio(work, vhash);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					quarkhash(vhash, endiandata);
					// bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			/*
			else if (vhash[7] > ptarget[7]) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
			*/
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (true);

	return 0;
}

// cleanup
extern "C" void free_quark()
{
	if (!init_flag)
		return;

	hipDeviceSynchronize();

	hipFree(d_hash);

	// if (cuda_arch[dev_id] >= 300) {
		hipFree(d_branch1Nonces);
		hipFree(d_branch2Nonces);
		hipFree(d_branch3Nonces);
	/*
	} else {
		hipFree(d_hash_br2[thr_id]);
	}
	*/

	quark_blake512_cpu_free();
	quark_groestl512_cpu_free();
	// quark_compactTest_cpu_free();

	cuda_check_cpu_free();
	init_flag = false;

	hipDeviceSynchronize();
}
