#include "hip/hip_runtime.h"
/**
 * JHA v8 algorithm - compatible implementation
 * @author tpruvot@github 05-2017
 */

extern "C" {
#include "sph/sph_keccak.h"
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_jh.h"
#include "sph/sph_skein.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include "quark/cuda_quark.h"
#include "cuda_debug.cuh"

static uint32_t *d_hash = nullptr;
static uint32_t *d_hash_br2;
static uint32_t *d_tempBranch;

extern void jackpot_keccak512_cpu_init(uint32_t threads);
extern void jackpot_keccak512_cpu_setBlock(void *pdata, size_t inlen);
extern void jackpot_keccak512_cpu_hash(uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);

// CPU HASH
extern "C" void jha_hash(void *output, const void *input)
{
	uint32_t hash[16];

	sph_blake512_context     ctx_blake;
	sph_groestl512_context   ctx_groestl;
	sph_jh512_context        ctx_jh;
	sph_keccak512_context    ctx_keccak;
	sph_skein512_context     ctx_skein;

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512 (&ctx_keccak, input, 80);
	sph_keccak512_close(&ctx_keccak, hash);

	for (int rnd = 0; rnd < 3; rnd++)
	{
		if (hash[0] & 0x01) {
			sph_groestl512_init(&ctx_groestl);
			sph_groestl512 (&ctx_groestl, (&hash), 64);
			sph_groestl512_close(&ctx_groestl, (&hash));
		}
		else {
			sph_skein512_init(&ctx_skein);
			sph_skein512 (&ctx_skein, (&hash), 64);
			sph_skein512_close(&ctx_skein, (&hash));
		}

		if (hash[0] & 0x01) {
			sph_blake512_init(&ctx_blake);
			sph_blake512 (&ctx_blake, (&hash), 64);
			sph_blake512_close(&ctx_blake, (&hash));
		}
		else {
			sph_jh512_init(&ctx_jh);
			sph_jh512 (&ctx_jh, (&hash), 64);
			sph_jh512_close(&ctx_jh, (&hash));
		}
	}
	memcpy(output, hash, 32);
}

__global__ __launch_bounds__(128, 8)
void jha_filter_gpu(const uint32_t threads, const uint32_t* d_hash, uint32_t* d_branch2, uint32_t* d_NonceBranch)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t offset = thread * 16U; // 64U / sizeof(uint32_t);
		uint4 *psrc = (uint4*) (&d_hash[offset]);
		d_NonceBranch[thread] = ((uint8_t*)psrc)[0] & 0x01;
		if (d_NonceBranch[thread]) return;
		// uint4 = 4x uint32_t = 16 bytes
		uint4 *pdst = (uint4*) (&d_branch2[offset]);
		pdst[0] = psrc[0];
		pdst[1] = psrc[1];
		pdst[2] = psrc[2];
		pdst[3] = psrc[3];
	}
}

__global__ __launch_bounds__(128, 8)
void jha_merge_gpu(const uint32_t threads, uint32_t* d_hash, uint32_t* d_branch2, uint32_t* const d_NonceBranch)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads && !d_NonceBranch[thread])
	{
		const uint32_t offset = thread * 16U;
		uint4 *pdst = (uint4*) (&d_hash[offset]);
		uint4 *psrc = (uint4*) (&d_branch2[offset]);
		pdst[0] = psrc[0];
		pdst[1] = psrc[1];
		pdst[2] = psrc[2];
		pdst[3] = psrc[3];
	}
}

__host__
uint32_t jha_filter_cpu(const uint32_t threads, const uint32_t *inpHashes, uint32_t* d_branch2)
{
	const uint32_t threadsperblock = 128;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);
	// extract algo permution hashes to a second branch buffer
	jha_filter_gpu <<<grid, block>>> (threads, inpHashes, d_branch2, d_tempBranch);
	return threads;
}

__host__
void jha_merge_cpu(const uint32_t threads, uint32_t *outpHashes, uint32_t* d_branch2)
{
	const uint32_t threadsperblock = 128;
	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);
	// put back second branch hashes to the common buffer d_hash
	jha_merge_gpu <<<grid, block>>> (threads, outpHashes, d_branch2, d_tempBranch);
}

static bool init_flag = false;

extern "C" int scanhash_jha(struct work *work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t __align__(64) endiandata[22];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;

	const uint32_t first_nonce = pdata[19];
	int dev_id = 0;

	uint32_t throughput =  1U << 10;
	if (init_flag) throughput = min(throughput, max_nonce - first_nonce);

	// if (opt_benchmark)
	ptarget[7] = 0x000f;

	if (!init_flag)
	{
		hipSetDevice(dev_id);
		/*
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		cuda_get_arch(thr_id);
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);
		*/
		CUDA_SAFE_CALL(hipMalloc(&d_hash, (size_t) 64 * throughput));
		hipMemset(d_hash, 0, (size_t) 64 * throughput);
		CUDA_SAFE_CALL(hipMalloc(&d_hash_br2, (size_t) 64 * throughput));
		CUDA_SAFE_CALL(hipMalloc(&d_tempBranch, sizeof(uint32_t) * throughput));

		jackpot_keccak512_cpu_init(throughput);
		quark_blake512_cpu_init(throughput);
		quark_groestl512_cpu_init(throughput);
		quark_jh512_cpu_init(throughput);
		quark_skein512_cpu_init(throughput);

		cuda_check_cpu_init(throughput);

		init_flag = true;
	}

	for (int k=0; k < 22; k++)
		be32enc(&endiandata[k], pdata[k]);

	jackpot_keccak512_cpu_setBlock((void*)endiandata, 80);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;

		jackpot_keccak512_cpu_hash(throughput, pdata[19], d_hash, order++);

		/*
		hipMemcpy(h_hash, d_hash, (size_t) 64 * throughput, hipMemcpyDeviceToHost);
		sprintf(buffer, "d_hash%d", file_num++);
		fp = fopen(buffer, "w+");
		for(int i = 0; i < (64 * throughput)/(sizeof(uint32_t)); i++) {
			fprintf(fp, "%u, ", h_hash[i]);
		}
		*/

		for (int rnd = 0; rnd < 3; rnd++)
		{
			jha_filter_cpu(throughput, d_hash, d_hash_br2);
			quark_groestl512_cpu_hash_64(throughput, pdata[19], NULL, d_hash, order++);
			quark_skein512_cpu_hash_64(throughput, pdata[19], NULL, d_hash_br2, order++);
			jha_merge_cpu(throughput, d_hash, d_hash_br2);

			jha_filter_cpu(throughput, d_hash, d_hash_br2);
			quark_blake512_cpu_hash_64(throughput, pdata[19], NULL, d_hash, order++);
			quark_jh512_cpu_hash_64(throughput, pdata[19], NULL, d_hash_br2, order++);
			jha_merge_cpu(throughput, d_hash, d_hash_br2);
		}

		*hashes_done = pdata[19] - first_nonce + throughput;

		// CUDA_LOG_ERROR()		
		work->nonces[0] = cuda_check_hash(throughput, pdata[19], d_hash);

		if (work->nonces[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t __align__(64) vhash[8];

			be32enc(&endiandata[19], work->nonces[0]);
			jha_hash(vhash, endiandata);

			if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				// work_set_target_ratio(work, vhash);
				work->nonces[1] = cuda_check_hash_suppl(throughput, pdata[19], d_hash, 1);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					jha_hash(vhash, endiandata);
					// bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				/*
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				*/
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (true);

	*hashes_done = pdata[19] - first_nonce;

	// CUDA_LOG_ERROR();

	return 0;
}

// cleanup
extern "C" void free_jha()
{
	if (!init_flag)
		return;

	hipDeviceSynchronize();

	hipFree(d_hash);
	hipFree(d_hash_br2);
	hipFree(d_tempBranch);

	quark_blake512_cpu_free();
	quark_groestl512_cpu_free();

	cuda_check_cpu_free();
	// CUDA_LOG_ERROR();

	hipDeviceSynchronize();
	init_flag = false;
}
