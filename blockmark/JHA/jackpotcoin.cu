#include "hip/hip_runtime.h"
extern "C"
{
#include "sph/sph_keccak.h"
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_jh.h"
#include "sph/sph_skein.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include "quark/cuda_quark.h"

static uint32_t *d_hash = nullptr;

// Speicher zur Generierung der Noncevektoren für die bedingten Hashes
static uint32_t *d_jackpotNonces = nullptr;
static uint32_t *d_branch1Nonces = nullptr;
static uint32_t *d_branch2Nonces = nullptr;
static uint32_t *d_branch3Nonces = nullptr;

extern void jackpot_keccak512_cpu_init(uint32_t threads);
extern void jackpot_keccak512_cpu_setBlock(void *pdata, size_t inlen);
extern void jackpot_keccak512_cpu_hash(uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);

/*
extern void jackpot_compactTest_cpu_init(uint32_t threads);
extern void jackpot_compactTest_cpu_free();
extern void jackpot_compactTest_cpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *inpHashes, uint32_t *d_validNonceTable,
                                            uint32_t *d_nonces1, uint32_t *nrm1, uint32_t *d_nonces2, uint32_t *nrm2, int order);
*/
extern uint32_t cuda_check_hash_branch(uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_inputHash, int order);

// CPU HASH JHA v8
extern "C" void jackpothash(void *state, const void *input)
{
	uint32_t hash[16];
	unsigned int rnd;

	sph_blake512_context     ctx_blake;
	sph_groestl512_context   ctx_groestl;
	sph_jh512_context        ctx_jh;
	sph_keccak512_context    ctx_keccak;
	sph_skein512_context     ctx_skein;

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512 (&ctx_keccak, input, 80);
	sph_keccak512_close(&ctx_keccak, hash);

	for (rnd = 0; rnd < 3; rnd++)
	{
		if (hash[0] & 0x01) {
			sph_groestl512_init(&ctx_groestl);
			sph_groestl512 (&ctx_groestl, (&hash), 64);
			sph_groestl512_close(&ctx_groestl, (&hash));
		}
		else {
			sph_skein512_init(&ctx_skein);
			sph_skein512 (&ctx_skein, (&hash), 64);
			sph_skein512_close(&ctx_skein, (&hash));
		}

		if (hash[0] & 0x01) {
			sph_blake512_init(&ctx_blake);
			sph_blake512 (&ctx_blake, (&hash), 64);
			sph_blake512_close(&ctx_blake, (&hash));
		}
		else {
			sph_jh512_init(&ctx_jh);
			sph_jh512 (&ctx_jh, (&hash), 64);
			sph_jh512_close(&ctx_jh, (&hash));
		}
	}
	memcpy(state, hash, 32);
}

static bool init_flag = false;

extern "C" int scanhash_jackpot(struct work *work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t __align__(64) endiandata[22];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int dev_id = 0;

	uint32_t throughput =  1U << 10;
	if (init_flag) throughput = min(throughput, max_nonce - first_nonce);

	// if (opt_benchmark)
		ptarget[7] = 0x000f;

	if (!init_flag)
	{
		hipSetDevice(dev_id);
		/*
		if (device_sm[dev_id] < 300 || cuda_arch[dev_id] < 300) {
			gpulog(LOG_ERR, thr_id, "Sorry, This algo is not supported by this GPU arch (SM 3.0 required)");
			proper_exit(EXIT_CODE_CUDA_ERROR);
		}

		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);
		*/
		CUDA_SAFE_CALL(hipMalloc(&d_hash, (size_t) 64 * throughput));

		jackpot_keccak512_cpu_init(throughput);
		// jackpot_compactTest_cpu_init(throughput);
		quark_blake512_cpu_init(throughput);
		quark_groestl512_cpu_init(throughput);
		quark_jh512_cpu_init(throughput);
		quark_skein512_cpu_init(throughput);

		cuda_check_cpu_init(throughput);

		hipMalloc(&d_branch1Nonces, (size_t) sizeof(uint32_t)*throughput*2);
		hipMalloc(&d_branch2Nonces, (size_t) sizeof(uint32_t)*throughput*2);
		hipMalloc(&d_branch3Nonces, (size_t) sizeof(uint32_t)*throughput*2);

		CUDA_SAFE_CALL(hipMalloc(&d_jackpotNonces, (size_t) sizeof(uint32_t)*throughput*2));

		init_flag = true;
	}

	for (int k=0; k < 22; k++)
		be32enc(&endiandata[k], pdata[k]);

	jackpot_keccak512_cpu_setBlock((void*)endiandata, 80);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;

		// erstes Keccak512 Hash mit CUDA
		jackpot_keccak512_cpu_hash(throughput, pdata[19], d_hash, order++);

		uint32_t nrm1, nrm2, nrm3;

		// Runde 1 (ohne Gröstl)

		/* jackpot_compactTest_cpu_hash_64(throughput, pdata[19], d_hash, NULL,
				d_branch1Nonces, &nrm1,
				d_branch3Nonces, &nrm3,
				order++); */

		// verfolge den skein-pfad weiter
		quark_skein512_cpu_hash_64(nrm3, pdata[19], d_branch3Nonces, d_hash, order++);

		// noch schnell Blake & JH
		/* jackpot_compactTest_cpu_hash_64(nrm3, pdata[19], d_hash, d_branch3Nonces,
			d_branch1Nonces, &nrm1,
			d_branch2Nonces, &nrm2,
			order++); */

		if (nrm1+nrm2 == nrm3) {
			quark_blake512_cpu_hash_64(nrm1, pdata[19], d_branch1Nonces, d_hash, order++);
			quark_jh512_cpu_hash_64(nrm2, pdata[19], d_branch2Nonces, d_hash, order++);
		}

		// Runde 3 (komplett)

		// jackpotNonces in branch1/2 aufsplitten gemäss if (hash[0] & 0x01)
		/* jackpot_compactTest_cpu_hash_64(nrm3, pdata[19], d_hash, d_branch3Nonces,
			d_branch1Nonces, &nrm1,
			d_branch2Nonces, &nrm2,
			order++); */

		if (nrm1+nrm2 == nrm3) {
			quark_groestl512_cpu_hash_64(nrm1, pdata[19], d_branch1Nonces, d_hash, order++);
			quark_skein512_cpu_hash_64(nrm2, pdata[19], d_branch2Nonces, d_hash, order++);
		}

		// jackpotNonces in branch1/2 aufsplitten gemäss if (hash[0] & 0x01)
		/* jackpot_compactTest_cpu_hash_64(nrm3, pdata[19], d_hash, d_branch3Nonces,
			d_branch1Nonces, &nrm1,
			d_branch2Nonces, &nrm2,
			order++); */

		if (nrm1+nrm2 == nrm3) {
			quark_blake512_cpu_hash_64(nrm1, pdata[19], d_branch1Nonces, d_hash, order++);
			quark_jh512_cpu_hash_64(nrm2, pdata[19], d_branch2Nonces, d_hash, order++);
		}

		// Runde 3 (komplett)

		// jackpotNonces in branch1/2 aufsplitten gemäss if (hash[0] & 0x01)
		/* jackpot_compactTest_cpu_hash_64(nrm3, pdata[19], d_hash, d_branch3Nonces,
			d_branch1Nonces, &nrm1,
			d_branch2Nonces, &nrm2,
			order++); */

		if (nrm1+nrm2 == nrm3) {
			quark_groestl512_cpu_hash_64(nrm1, pdata[19], d_branch1Nonces, d_hash, order++);
			quark_skein512_cpu_hash_64(nrm2, pdata[19], d_branch2Nonces, d_hash, order++);
		}

		// jackpotNonces in branch1/2 aufsplitten gemäss if (hash[0] & 0x01)
		/* jackpot_compactTest_cpu_hash_64(nrm3, pdata[19], d_hash, d_branch3Nonces,
			d_branch1Nonces, &nrm1,
			d_branch2Nonces, &nrm2,
			order++); */

		if (nrm1+nrm2 == nrm3) {
			quark_blake512_cpu_hash_64(nrm1, pdata[19], d_branch1Nonces, d_hash, order++);
			quark_jh512_cpu_hash_64(nrm2, pdata[19], d_branch2Nonces, d_hash, order++);
		}

		*hashes_done = pdata[19] - first_nonce + throughput;

		// CUDA_LOG_ERROR();

		work->nonces[0] = cuda_check_hash_branch(nrm3, pdata[19], d_branch3Nonces, d_hash, order++);

		if (work->nonces[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t __align__(64) vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);

			// jackpothash function gibt die Zahl der Runden zurück
			jackpothash(vhash, endiandata);

			if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				// work_set_target_ratio(work, vhash);
#if 0
				work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					jackpothash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
#else
				pdata[19] = work->nonces[0] + 1; // cursor
#endif
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				/*
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				*/
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (true);

	*hashes_done = pdata[19] - first_nonce;

	// CUDA_LOG_ERROR();

	return 0;
}

// cleanup
extern "C" void free_jackpot()
{
	if (!init_flag)
		return;

	hipDeviceSynchronize();

	hipFree(d_branch1Nonces);
	hipFree(d_branch2Nonces);
	hipFree(d_branch3Nonces);
	hipFree(d_jackpotNonces);

	quark_blake512_cpu_free();
	quark_groestl512_cpu_free();
	// jackpot_compactTest_cpu_free();

	hipFree(d_hash);

	cuda_check_cpu_free();
	// CUDA_LOG_ERROR();

	hipDeviceSynchronize();

	init_flag = false;
}
